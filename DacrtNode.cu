#include "hip/hip_runtime.h"
// DACRT node
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <DacrtNode.h>

#include <Cone.h>
#include <ForEachWithOwners.h>
#include <HyperCube.h>
#include <HyperRays.h>
#include <RayContainer.h>
#include <SphereGeometry.h>
#include <SphereContainer.h>
#include <ToString.h>

#include <sstream>

#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_scan.h>
#include <thrust/transform.h>

// *** DacrtNode ***

std::string DacrtNode::ToString() const {
    std::ostringstream out;
    out << "[rays: [" << rayStart << " -> " << rayEnd << "], spheres: [" << sphereStart << " -> " << sphereEnd << "]]";
    return out.str();
}

// *** DacrtNodes ***

DacrtNodes::DacrtNodes(const size_t capacity) 
    : scan1(capacity+1), scan2(capacity+1), 
      rayPartitions(capacity), spherePartitions(capacity),
      nextRayPartitions(capacity), nextSpherePartitions(capacity),
      doneRayPartitions(capacity), doneSpherePartitions(capacity) {
    scan1[0] = scan2[0] = 0;
    rayPartitions.resize(0); spherePartitions.resize(0);
    nextRayPartitions.resize(0); nextSpherePartitions.resize(0);
    doneRayPartitions.resize(0); doneSpherePartitions.resize(0);
}

void DacrtNodes::Reset() {
    rayPartitions.resize(0);
    nextRayPartitions.resize(0);
    spherePartitions.resize(0);
    nextSpherePartitions.resize(0);
    doneRayPartitions.resize(0);
    doneSpherePartitions.resize(0);
}

struct CalcSplitInfo {
    __host__ __device__
    thrust::tuple<Axis, float> operator()(thrust::tuple<float2, float2, float2, float2, float2> val) {
        float2 x = thrust::get<0>(val);
        float range = x.y - x.x;
        Axis axis = X;
        float split = (x.y + x.x) * 0.5f;
        
        float2 y = thrust::get<1>(val);
        float yRange = y.y - y.x;
        if (range < yRange) {
            axis = Y;
            split = (y.y + y.x) * 0.5f;
        }
        
        float2 z = thrust::get<2>(val);
        float zRange = z.y - z.x;
        if (range < zRange) {
            axis = Z;
            split = (z.y + z.x) * 0.5f;
        }
        
        float2 u = thrust::get<3>(val);
        float uRange = u.y - u.x;
        if (range < uRange) {
            axis = U;
            split = (u.y + u.x) * 0.5f;
        }

        float2 v = thrust::get<4>(val);
        float vRange = v.y - v.x;
        if (range < vRange) {
            axis = V;
            split = (v.y + v.x) * 0.5f;
        }
        
        return thrust::tuple<Axis, float>(axis, split);
    }
};

struct RayPartitionSide {
    float4 *rayOrigins, *rayAxisUVs;
    
    Axis *splitAxis;
    float *splitValues;
    
    // result
    PartitionSide *partitionSides;
    
    RayPartitionSide(thrust::device_vector<Axis>& axis, thrust::device_vector<float>& values)
        : splitAxis(thrust::raw_pointer_cast(axis.data())), 
          splitValues(thrust::raw_pointer_cast(values.data())) {}

    RayPartitionSide(HyperRays::Iterator rays,
                     thrust::device_vector<Axis>& axis, thrust::device_vector<float>& values,
                     thrust::device_vector<PartitionSide>& sides)
        : rayOrigins(RawPointer(HyperRays::GetOrigins(rays))),
          rayAxisUVs(RawPointer(HyperRays::GetAxisUVs(rays))),
          splitAxis(RawPointer(axis)), 
          splitValues(RawPointer(values)),
          partitionSides(RawPointer(sides)) {}
    
    __host__ __device__
    PartitionSide operator()(thrust::tuple<thrust::tuple<float4, float4>, unsigned int> ray) {
        int owner = thrust::get<1>(ray);
        Axis axis = splitAxis[owner];
        float splitVal = splitValues[owner];
        
        float rayVals[5];
        float4 origin = thrust::get<0>(thrust::get<0>(ray));
        rayVals[0] = origin.x;
        rayVals[1] = origin.y;
        rayVals[2] = origin.z;
        
        float4 UV = thrust::get<1>(thrust::get<0>(ray));
        rayVals[3] = UV.y;
        rayVals[4] = UV.z;
        
        return rayVals[axis] <= splitVal ? LEFT : RIGHT;
    }
    
    __host__ __device__
    void operator()(const unsigned int index, const unsigned int owner) const {
        const Axis axis = splitAxis[owner];
        const float splitVal = splitValues[owner];

        // IDEA since most owners will cover a warp or more, perhaps it will be
        // slightly faster to branch on (axis < 3) and avoid a memory lookup?
        // Only ever so slightly though.
        float rayVals[5];
        const float3 origin = make_float3(rayOrigins[index]);
        rayVals[0] = origin.x;
        rayVals[1] = origin.y;
        rayVals[2] = origin.z;
        
        const float3 axisUV = make_float3(rayAxisUVs[index]);
        rayVals[3] = axisUV.y;
        rayVals[4] = axisUV.z;
        
        partitionSides[index]  = rayVals[axis] <= splitVal ? LEFT : RIGHT;
    }
    
};

template <int S>
struct SideToOne {
    __host__ __device__ unsigned int operator()(PartitionSide s) { return s & S ? 1 : 0; }
};
static SideToOne<LEFT> leftToOne;
static SideToOne<RIGHT> rightToOne;
static thrust::plus<unsigned int> plus;
struct BoolToInt { __host__ __device__ unsigned int operator()(bool b) { return (int)b; } };

struct CreateCones {
    __host__ __device__
    Cone operator()(const thrust::tuple<SignedAxis, float2, float2, float2, float2, float2> c) const {
        const HyperCube cube(thrust::get<0>(c), thrust::get<1>(c), thrust::get<2>(c),
                             thrust::get<3>(c), thrust::get<4>(c), thrust::get<5>(c));
        
        return Cone::FromCube(cube);
    }
};
static CreateCones createCones;

__constant__ unsigned int d_oldCubeCount;

struct CubesFromSplitPlanes {
    SignedAxis* a;
    float2 *x, *y, *z, *u, *v;
    Axis* splitAxis;
    float* splitValues;
    
    CubesFromSplitPlanes(HyperCubes& cubes, thrust::device_vector<Axis>& sAxis,
                         thrust::device_vector<float>& sValues) 
        : a(thrust::raw_pointer_cast(cubes.a.data())), 
          x(thrust::raw_pointer_cast(cubes.x.data())), 
          y(thrust::raw_pointer_cast(cubes.y.data())), 
          z(thrust::raw_pointer_cast(cubes.z.data())), 
          u(thrust::raw_pointer_cast(cubes.u.data())), 
          v(thrust::raw_pointer_cast(cubes.v.data())),
          splitAxis(thrust::raw_pointer_cast(sAxis.data())),
          splitValues(thrust::raw_pointer_cast(sValues.data())) {
        unsigned int oldCubeCount = cubes.Size();
        hipMemcpyToSymbol(HIP_SYMBOL(d_oldCubeCount), &oldCubeCount, sizeof(unsigned int));
    }
    
    __host__ __device__
    thrust::tuple<SignedAxis, float2, float2, float2, float2, float2> operator()(const unsigned int threadId) const {
        const unsigned int oldCubeId = threadId % d_oldCubeCount;
        const PartitionSide side = threadId < d_oldCubeCount ? LEFT : RIGHT;
        const Axis sAxis = splitAxis[oldCubeId];
        const float splitValue = splitValues[oldCubeId];
        return thrust::tuple<SignedAxis, float2, float2, float2, float2, float2>
            (a[oldCubeId], 
             CalcBounds(sAxis == X, side, x[oldCubeId], splitValue),
             CalcBounds(sAxis == Y, side, y[oldCubeId], splitValue),
             CalcBounds(sAxis == Z, side, z[oldCubeId], splitValue),
             CalcBounds(sAxis == U, side, u[oldCubeId], splitValue),
             CalcBounds(sAxis == V, side, v[oldCubeId], splitValue));
    }
    __host__ __device__
    inline float2 CalcBounds(const bool split, const PartitionSide side, const float2 bounds, const float splitVal) const {
        return split ? make_float2(side == LEFT ? bounds.x : splitVal,
                                   side == RIGHT ? bounds.y : splitVal) : bounds;
    }
};

struct SpherePartitioningByCones {
    Cone* cones;
    Sphere* spheres;
    SpherePartitioningByCones(thrust::device_vector<Cone>& cs, 
                              thrust::device_vector<Sphere>& ss)
        : cones(thrust::raw_pointer_cast(cs.data())),
          spheres(thrust::raw_pointer_cast(ss.data())) {}
    
    __device__
    PartitionSide operator()(const unsigned int sphereId, const unsigned int owner) const {
        const Sphere sphere = spheres[sphereId];
        
        const Cone leftCone = cones[owner];
        PartitionSide side = leftCone.DoesIntersect(sphere) ? LEFT : NONE;
        
        const Cone rightCone = cones[owner + d_oldCubeCount];
        return (PartitionSide)(side | (rightCone.DoesIntersect(sphere) ? RIGHT : NONE));
    }
};

__constant__ unsigned int d_spheresMovedLeft;
struct AddSpheresMovedLeft {
    AddSpheresMovedLeft(thrust::device_vector<unsigned int>& leftIndices){
        unsigned int* spheresMovedLeft = thrust::raw_pointer_cast(leftIndices.data()) + leftIndices.size()-1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_spheresMovedLeft), spheresMovedLeft, sizeof(unsigned int), 0, hipMemcpyDeviceToDevice);
    }
    __device__
    unsigned int operator()(const unsigned int v) const {
        return v + d_spheresMovedLeft;
    }
};

__constant__ unsigned int d_raysMovedLeft;

struct ComputeNewNodePartitions {
    unsigned int* rayLeftIndices;
    unsigned int *sphereLeftIndices, *sphereRightIndices;
    uint2 *rayPartitions, *spherePartitions;
    
    ComputeNewNodePartitions(thrust::device_vector<unsigned int>& rLeftIndices,
                             thrust::device_vector<unsigned int>& sLeftIndices,
                             thrust::device_vector<unsigned int>& sRightIndices)
        : rayLeftIndices(thrust::raw_pointer_cast(rLeftIndices.data())),
          sphereLeftIndices(thrust::raw_pointer_cast(sLeftIndices.data())),
          sphereRightIndices(thrust::raw_pointer_cast(sRightIndices.data())) {
        unsigned int* data = thrust::raw_pointer_cast(rLeftIndices.data()) + rLeftIndices.size()-1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_raysMovedLeft), data, sizeof(unsigned int), 0, hipMemcpyDeviceToDevice);
    }

    __device__
    thrust::tuple<uint4, uint4> operator()(const uint2 rayPartition, const uint2 spherePartition) const {
        uint4 rays;
        const unsigned int rBegin = rays.x = rayLeftIndices[rayPartition.x];
        const unsigned int rEnd = rays.y = rayLeftIndices[rayPartition.y];
        rays.z = rayPartition.x - rBegin + d_raysMovedLeft;
        rays.w = rayPartition.y - rEnd + d_raysMovedLeft;

        uint4 sphere;
        sphere.x = sphereLeftIndices[spherePartition.x];
        sphere.y = sphereLeftIndices[spherePartition.y];
        sphere.z = sphereRightIndices[spherePartition.x];
        sphere.w = sphereRightIndices[spherePartition.y];
        
        return thrust::tuple<uint4, uint4>(rays, sphere);
    }

};

struct ComputeNewLeftNodePartitions {
    unsigned int* rayLeftIndices;
    unsigned int* sphereLeftIndices;
    unsigned int* sphereRightIndices;
    
    ComputeNewLeftNodePartitions(thrust::device_vector<unsigned int>& rLeftIndices,
                                 thrust::device_vector<unsigned int>& sLeftIndices,
                                 thrust::device_vector<unsigned int>& sRightIndices)
        : rayLeftIndices(thrust::raw_pointer_cast(rLeftIndices.data())),
          sphereLeftIndices(thrust::raw_pointer_cast(sLeftIndices.data())),
          sphereRightIndices(thrust::raw_pointer_cast(sRightIndices.data())) {
        unsigned int* data = thrust::raw_pointer_cast(rLeftIndices.data()) + rLeftIndices.size()-1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_raysMovedLeft), data, sizeof(unsigned int), 0, hipMemcpyDeviceToDevice);
    }
    __device__
    thrust::tuple<uint2, uint2> operator()(const uint2 rayPartition, const uint2 spherePartition) const {
        uint2 rays;
        rays.x = rayLeftIndices[rayPartition.x];
        rays.y = rayLeftIndices[rayPartition.y];

        uint2 sphere;
        sphere.x = sphereLeftIndices[spherePartition.x];
        sphere.y = sphereLeftIndices[spherePartition.y];
        
        return thrust::tuple<uint2, uint2>(rays, sphere);
    }
    
};

struct ComputeNewRightNodePartitions {
    unsigned int* rayLeftIndices;
    unsigned int* sphereLeftIndices;
    unsigned int* sphereRightIndices;
    
    ComputeNewRightNodePartitions(thrust::device_vector<unsigned int>& rLeftIndices,
                                  thrust::device_vector<unsigned int>& sLeftIndices,
                                  thrust::device_vector<unsigned int>& sRightIndices)
        : rayLeftIndices(thrust::raw_pointer_cast(rLeftIndices.data())),
          sphereLeftIndices(thrust::raw_pointer_cast(sLeftIndices.data())),
          sphereRightIndices(thrust::raw_pointer_cast(sRightIndices.data())) {
        unsigned int* data = thrust::raw_pointer_cast(rLeftIndices.data()) + rLeftIndices.size()-1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_raysMovedLeft), data, sizeof(unsigned int), 0, hipMemcpyDeviceToDevice);
    }
    __device__
    thrust::tuple<uint2, uint2> operator()(const uint2 rayPartition, const uint2 spherePartition) const {
        uint2 rays;
        const unsigned int rBegin = rayLeftIndices[rayPartition.x];
        const unsigned int rEnd = rayLeftIndices[rayPartition.y];
        rays.x = rayPartition.x - rBegin + d_raysMovedLeft;
        rays.y = rayPartition.y - rEnd + d_raysMovedLeft;

        uint2 sphere;
        sphere.x = sphereRightIndices[spherePartition.x];
        sphere.y = sphereRightIndices[spherePartition.y];
        
        return thrust::tuple<uint2, uint2>(rays, sphere);
    }
};

void DacrtNodes::Partition(RayContainer& rays, SphereContainer& spheres,
                           HyperCubes& cubes) {

    // TODO move static left and right indices vectors to global scope? Do I
    // need more than one at a time? 
    /// No! But I need to split next ray and sphere partition creation.

    size_t rayCount = rays.InnerSize();

    // Calculate splitting info
    static thrust::device_vector<Axis> splitAxis(cubes.Size());
    splitAxis.resize(cubes.Size());
    static thrust::device_vector<float> splitValues(cubes.Size());
    splitValues.resize(cubes.Size());
    thrust::zip_iterator<thrust::tuple<AxisIterator, FloatIterator> > axisInfo
        = thrust::make_zip_iterator(thrust::make_tuple(splitAxis.begin(), splitValues.begin()));

    CalcSplitInfo calcSplitInfo;    
    thrust::transform(cubes.BeginBounds(), cubes.EndBounds(), axisInfo, calcSplitInfo);    

    // Calculate the partition side
    static thrust::device_vector<PartitionSide> rayPartitionSides(rayCount);
    rayPartitionSides.resize(rayCount);

    // Calculate current ray owners.
#if 0
    static thrust::device_vector<unsigned int> rayOwners(rayCount);
    rayOwners.resize(rayCount);
    CalcOwners(rayPartitions, rayOwners);
    thrust::zip_iterator<thrust::tuple<HyperRays::Iterator, UintIterator> > raysWithOwners
        = thrust::make_zip_iterator(thrust::make_tuple(rays.BeginInnerRays(), rayOwners.begin()));

    RayPartitionSide rayPartitionSide = RayPartitionSide(splitAxis, splitValues);
    thrust::transform(raysWithOwners, raysWithOwners + rayCount, 
                      rayPartitionSides.begin(), rayPartitionSide);
#else
    RayPartitionSide rayPartitionSide = RayPartitionSide(rays.BeginInnerRays(), splitAxis, splitValues,
                                                         rayPartitionSides);
    ForEachWithOwners(rayPartitions, 0, rayPartitions.size(), 
                      rayCount, rayPartitionSide);
#endif
    
    // Calculate the indices for the rays moved left using scan
    static thrust::device_vector<unsigned int> rayLeftIndices(rayCount+1);
    rayLeftIndices.resize(rayCount+1);
    rayLeftIndices[0] = 0; // Should be handled by resize not being destructive.
    thrust::transform_inclusive_scan(rayPartitionSides.begin(), rayPartitionSides.end(),
                                     rayLeftIndices.begin()+1, leftToOne, plus);
    
    // Scatter the rays
    rays.Partition(rayPartitionSides, rayLeftIndices);

    // Calculate the new hypercubes 
    /// IDEA: Since the rays have been scattered, just reduce them, but that
    // would mean also scattering the hypercubes when creating leaves.
    static HyperCubes splitCubes(cubes.Size() * 2);
    splitCubes.Resize(cubes.Size() * 2);
    CubesFromSplitPlanes cubesFromSplitPlanes(cubes, splitAxis, splitValues);
    thrust::transform(thrust::counting_iterator<unsigned int>(0), thrust::counting_iterator<unsigned int>(cubes.Size() * 2), 
                      splitCubes.Begin(), cubesFromSplitPlanes);
    
    // Calculate the cones used for splitting 
    // TODO using knowledge about the cube split, the resulting two cones can be
    // computed faster if computed together in one thread.
    static thrust::device_vector<Cone> cones(cubes.Size());
    cones.resize(cubes.Size() * 2);
    thrust::transform(splitCubes.Begin(), splitCubes.End(), cones.begin(), createCones);

    // Calculate current sphere owners. TODO Use a work queue instead
    static thrust::device_vector<unsigned int> sphereOwners(spheres.CurrentSize());
    sphereOwners.resize(spheres.CurrentSize());
    CalcOwners(spherePartitions, sphereOwners);

    // Calculate sphere partitions
    static thrust::device_vector<PartitionSide> spherePartitionSides(spheres.CurrentSize());
    spherePartitionSides.resize(spheres.CurrentSize());
    SpherePartitioningByCones spherePartitioningByCones(cones, spheres.SphereGeometry().spheres);
    thrust::transform(spheres.BeginCurrentIndices(), spheres.EndCurrentIndices(), sphereOwners.begin(),
                      spherePartitionSides.begin(), spherePartitioningByCones);

    static thrust::device_vector<unsigned int> sphereLeftIndices(spheres.CurrentSize()+1);
    sphereLeftIndices.resize(spheres.CurrentSize()+1);
    sphereLeftIndices[0] = 0; // Should be handled by resize not being destructive.
    static thrust::device_vector<unsigned int> sphereRightIndices(spheres.CurrentSize()+1);
    sphereRightIndices.resize(spheres.CurrentSize()+1);
    sphereRightIndices[0] = 0; // Should be handled by resize not being destructive.
    
    thrust::transform_inclusive_scan(spherePartitionSides.begin(), spherePartitionSides.end(),
                                     sphereLeftIndices.begin()+1, leftToOne, plus);
    thrust::transform_inclusive_scan(spherePartitionSides.begin(), spherePartitionSides.end(),
                                     sphereRightIndices.begin()+1, rightToOne, plus);

    AddSpheresMovedLeft addSpheresMovedLeft(sphereLeftIndices);
    thrust::transform(sphereRightIndices.begin(), sphereRightIndices.end(), sphereRightIndices.begin(), addSpheresMovedLeft);

    // Scatter spheres
    spheres.Partition(spherePartitionSides, sphereLeftIndices, sphereRightIndices);
    
    // Compute new dacrt node partitions
    unsigned int nextUnfinishedNodes = UnfinishedNodes() * 2;
    nextRayPartitions.resize(nextUnfinishedNodes);
    nextSpherePartitions.resize(nextUnfinishedNodes);
    
    // Wrap partitions in uint4 to be able to store both left and right
    // simultaneously and coallesced. (Hackish)
    // thrust::device_ptr<uint4> nextRays((uint4*)(void*)thrust::raw_pointer_cast(nextRayPartitions.data()));
    // thrust::device_ptr<uint4> nextSpheres((uint4*)(void*)thrust::raw_pointer_cast(nextSpherePartitions.data()));
    
    // thrust::zip_iterator<thrust::tuple<thrust::device_ptr<uint4>, thrust::device_ptr<uint4> > > partitionWrapper = 
    //     thrust::make_zip_iterator(thrust::make_tuple(nextRays, nextSpheres));
    // ComputeNewNodePartitions computeNewNodePartitions(rayLeftIndices, sphereLeftIndices, sphereRightIndices);
    // thrust::transform(BeginUnfinishedRayPartitions(), EndUnfinishedRayPartitions(), BeginUnfinishedSpherePartitions(),
    //                   partitionWrapper, computeNewNodePartitions);

    thrust::zip_iterator<thrust::tuple<Uint2Iterator, Uint2Iterator > > partitionWrapper = 
        thrust::make_zip_iterator(thrust::make_tuple(nextRayPartitions.begin(), nextSpherePartitions.begin()));

    ComputeNewLeftNodePartitions computeNewLeftNodePartitions(rayLeftIndices, sphereLeftIndices, sphereRightIndices);
    thrust::transform(BeginUnfinishedRayPartitions(), EndUnfinishedRayPartitions(), BeginUnfinishedSpherePartitions(),
                      partitionWrapper, computeNewLeftNodePartitions);
    ComputeNewRightNodePartitions computeNewRightNodePartitions(rayLeftIndices, sphereLeftIndices, sphereRightIndices);
    thrust::transform(BeginUnfinishedRayPartitions(), EndUnfinishedRayPartitions(), BeginUnfinishedSpherePartitions(),
                      partitionWrapper+UnfinishedNodes(), computeNewRightNodePartitions);
    
    rayPartitions.swap(nextRayPartitions);
    spherePartitions.swap(nextSpherePartitions);
}


// *** LEAF PARTITIONING ***

struct IsNodeLeaf {
    __host__ __device__
    bool operator()(const uint2 rayPartition, const uint2 spherePartition) const {
        const float rayCount = (float)(rayPartition.y - rayPartition.x);
        const float sphereCount = (float)(spherePartition.y - spherePartition.x);
        
        return rayCount * sphereCount <= 16.0f * (rayCount + sphereCount);
    }
};

struct MarkLeafSize {
    __host__ __device__
    unsigned int operator()(const thrust::tuple<bool, uint2> input) const {
        bool isLeaf = thrust::get<0>(input);
        uint2 rayPartition = thrust::get<1>(input);
        return isLeaf ? rayPartition.y - rayPartition.x : 0;
    }
};

__constant__ unsigned int d_leafPartitionOffset;
struct NewPrimPartitions {
    uint2 *oldPartitions;
    unsigned int *leafIndices;
    bool *isLeafs;

    unsigned int *newBegins;
    uint2 *nextPartitions, *leafPartitions;

    NewPrimPartitions(thrust::device_vector<uint2>::iterator oPartitions,
                      thrust::device_vector<unsigned int>& lIndices,
                      thrust::device_vector<bool>& isLeafs,
                      thrust::device_vector<unsigned int>& nBegins,
                      thrust::device_vector<uint2>& nPartitions,
                      const unsigned int leafPartitionOffset, 
                      thrust::device_vector<uint2>& lPartitions,
                      const unsigned int leafOffset) 
        : oldPartitions(RawPointer(oPartitions)),
          leafIndices(RawPointer(lIndices)),
          isLeafs(RawPointer(isLeafs)),
          newBegins(RawPointer(nBegins)),
          nextPartitions(RawPointer(nPartitions)),
          leafPartitions(RawPointer(lPartitions) + leafOffset) {
        hipMemcpyToSymbol(HIP_SYMBOL(d_leafPartitionOffset), &leafPartitionOffset, sizeof(unsigned int));
    }
    
    __host__ __device__
    void operator()(const unsigned int threadId) const {
        const uint2 oldPartition = oldPartitions[threadId];
        const unsigned int range = oldPartition.y - oldPartition.x;
        const bool isLeaf = isLeafs[threadId];
        unsigned int newBegin = newBegins[oldPartition.x];
        newBegin += isLeaf ? d_leafPartitionOffset : 0;
        const uint2 partition = make_uint2(newBegin, newBegin + range);
        const unsigned int leafIndex = leafIndices[threadId];
        const unsigned int index = isLeaf ? leafIndex : threadId - leafIndex;
        uint2* output = isLeaf ? leafPartitions : nextPartitions;
        output[index] = partition;
    }
    
};

bool DacrtNodes::PartitionLeafs(RayContainer& rays, SphereContainer& spheres) {
    static thrust::device_vector<bool> isLeaf(UnfinishedNodes());
    isLeaf.resize(UnfinishedNodes());

    size_t unfinishedNodes = UnfinishedNodes();

    // TODO make isLeaf unsigned int and reuse for indices? isLeaf info is
    // stored in an index and it's neighbour.
    thrust::transform(BeginUnfinishedRayPartitions(), EndUnfinishedRayPartitions(), BeginUnfinishedSpherePartitions(),
                      isLeaf.begin(), IsNodeLeaf());
    // std::cout << "Leaf nodes:\n" << isLeaf << std::endl;

    static thrust::device_vector<unsigned int> leafIndices(UnfinishedNodes()+1);
    leafIndices.resize(unfinishedNodes+1);
    leafIndices[0] = 0;
    thrust::transform_inclusive_scan(isLeaf.begin(), isLeaf.end(), leafIndices.begin()+1, 
                                     BoolToInt(), plus);
    const unsigned int newLeafNodes = leafIndices[leafIndices.size()-1];
    const unsigned int oldLeafNodes = DoneNodes();

    if (newLeafNodes == 0) return false;

    // Partition rays
    static thrust::device_vector<unsigned int> rayLeafNodeIndices(unfinishedNodes+1); // TODO could be a globally static vector
    rayLeafNodeIndices.resize(unfinishedNodes+1);
    rayLeafNodeIndices[0] = 0;
    thrust::zip_iterator<thrust::tuple<BoolIterator, Uint2Iterator> > leafNodeValues =
        thrust::make_zip_iterator(thrust::make_tuple(isLeaf.begin(), BeginUnfinishedRayPartitions()));
    thrust::transform_inclusive_scan(leafNodeValues, leafNodeValues + unfinishedNodes, 
                                     rayLeafNodeIndices.begin()+1, MarkLeafSize(), plus);
    // std::cout << "Ray Leaf Node Indices:\n" << rayLeafNodeIndices << std::endl;

    static thrust::device_vector<unsigned int> owners(rays.InnerSize());
    owners.resize(rays.InnerSize());
    CalcOwners(rayPartitions, owners);
    
    const unsigned int oldRayLeafs = rays.LeafRays();
    rays.PartitionLeafs(isLeaf, rayLeafNodeIndices, rayPartitions, owners);
    // Owners now hold the new ray begin indices
    thrust::device_vector<unsigned int>& newRayIndices = owners;
    
    // New node ray partitions
    nextRayPartitions.resize(rayPartitions.size() - newLeafNodes);
    doneRayPartitions.resize(doneRayPartitions.size() + newLeafNodes);

    thrust::zip_iterator<thrust::tuple<Uint2Iterator, UintIterator, BoolIterator> > nodePartitionsInput =
        thrust::make_zip_iterator(thrust::make_tuple(BeginUnfinishedRayPartitions(), leafIndices.begin(), isLeaf.begin()));
    NewPrimPartitions newPrimPartitions(BeginUnfinishedRayPartitions(), leafIndices, isLeaf,
                                        newRayIndices, nextRayPartitions, oldRayLeafs, doneRayPartitions, oldLeafNodes);
    thrust::for_each(thrust::counting_iterator<unsigned int>(0), 
                     thrust::counting_iterator<unsigned int>(unfinishedNodes),
                     newPrimPartitions);
    
    rayPartitions.swap(nextRayPartitions);

    // Partition spheres
    static thrust::device_vector<unsigned int> sphereLeafNodeIndices(unfinishedNodes+1); // TODO could be a globally static vector
    sphereLeafNodeIndices.resize(unfinishedNodes+1);
    sphereLeafNodeIndices[0] = 0;
    leafNodeValues = thrust::make_zip_iterator(thrust::make_tuple(isLeaf.begin(), BeginUnfinishedSpherePartitions()));
    thrust::transform_inclusive_scan(leafNodeValues, leafNodeValues + unfinishedNodes, 
                                     sphereLeafNodeIndices.begin()+1, MarkLeafSize(), plus);

    owners.resize(spheres.CurrentSize());
    CalcOwners(spherePartitions, owners);

    const unsigned int oldSphereLeafs = spheres.DoneSize();
    spheres.PartitionLeafs(isLeaf, sphereLeafNodeIndices, spherePartitions, owners);

    // New node sphere partitions
    nextSpherePartitions.resize(spherePartitions.size() - newLeafNodes);
    doneSpherePartitions.resize(doneSpherePartitions.size() + newLeafNodes);
    nodePartitionsInput = thrust::make_zip_iterator(thrust::make_tuple(BeginUnfinishedSpherePartitions(), leafIndices.begin(), isLeaf.begin()));
    newPrimPartitions = NewPrimPartitions(BeginUnfinishedSpherePartitions(), leafIndices, isLeaf,
                                          owners, nextSpherePartitions, oldSphereLeafs, doneSpherePartitions, oldLeafNodes);
    thrust::for_each(thrust::counting_iterator<unsigned int>(0),
                     thrust::counting_iterator<unsigned int>(unfinishedNodes),
                     newPrimPartitions);
    
    spherePartitions.swap(nextSpherePartitions);

    return true;
}



// *** EXHAUSTIVE INTERSECTION ***

struct ExhaustiveIntersection {
    float4 *rayOrigins, *rayAxisUVs;
    uint2 *spherePartitions;
    unsigned int *sphereIndices;
    Sphere *spheres;

    unsigned int *hitIDs;
    
    ExhaustiveIntersection(HyperRays& rays, 
                           thrust::device_vector<uint2>& sPartitions,
                           thrust::device_vector<unsigned int>& sIndices, 
                           thrust::device_vector<Sphere>& ss,
                           thrust::device_vector<unsigned int>& hits)
        : rayOrigins(thrust::raw_pointer_cast(rays.origins.data())),
          rayAxisUVs(thrust::raw_pointer_cast(rays.axisUVs.data())),
          spherePartitions(thrust::raw_pointer_cast(sPartitions.data())), 
          sphereIndices(thrust::raw_pointer_cast(sIndices.data())), 
          spheres(thrust::raw_pointer_cast(ss.data())),
          hitIDs(thrust::raw_pointer_cast(hits.data()))
    {}

    /**
     * Takes a ray as argument and intersects it against all spheres referenced
     * by its parent DacrtNode.
     *
     * Returns the index of the intersected sphere and stores the distance to it
     * in the w component of the ray's direction.
     */
    __host__ __device__
    void operator()(const unsigned int index, const unsigned int owner) const {
        const float3 origin = make_float3(rayOrigins[index]);
        const float3 dir = normalize(HyperRay::AxisUVToDirection(make_float3(rayAxisUVs[index])));
        
        const uint2 spherePartition = spherePartitions[owner];
        float hitT = 1e30f;
        unsigned int hitID = SpheresGeometry::MISSED;
        
        for (unsigned int g = spherePartition.x; g < spherePartition.y; ++g) {
            const unsigned int sphereId = sphereIndices[g];
            const Sphere s = spheres[sphereId];
            const float t = s.Intersect(origin, dir);
            if (0 < t && t < hitT) {
                hitID = sphereId;
                hitT = t;
            }
        }

        rayAxisUVs[index] = make_float4(dir, hitT);
        hitIDs[index] = hitID;
    }
};

void DacrtNodes::ExhaustiveIntersect(RayContainer& rays, SphereContainer& spheres, 
                                     thrust::device_vector<unsigned int>& hits) {
    
    // std::cout << "ExhaustiveIntersect" << std::endl;
    hits.resize(rays.LeafRays());

    // std::cout << "doneRayPartitions:\n" << doneRayPartitions <<std::endl;

    ExhaustiveIntersection exhaustive(rays.leafRays, 
                                      doneSpherePartitions, 
                                      spheres.doneIndices, 
                                      spheres.spheres.spheres,
                                      hits);
    ForEachWithOwners(doneRayPartitions, 0, doneRayPartitions.size(),
                      hits.size(), exhaustive);

    //std::cout << "hits:\n" << hits << std::endl;
}

// *** CALC OWNERS ***

struct SetMarkers {
    unsigned int* owners;
    uint2* partitions;
    SetMarkers(thrust::device_vector<unsigned int>& owners,
               thrust::device_vector<uint2>& partitions)
        : owners(thrust::raw_pointer_cast(owners.data())),
          partitions(thrust::raw_pointer_cast(partitions.data())) {}
    
    __host__ __device__
    void operator()(const unsigned int threadId) const {
        const uint2 part = partitions[threadId];
        owners[part.x] = threadId == 0 ? 0 : 1;
    }
};

struct WriteOwner {
    unsigned int* owners;
    
    WriteOwner(thrust::device_vector<unsigned int>& os) 
        : owners(thrust::raw_pointer_cast(os.data())) {}

    __device__
    void operator()(const unsigned int index, const unsigned int owner) const {
        owners[index] = owner;
    }
};


void DacrtNodes::CalcOwners(thrust::device_vector<uint2>& partitions,
                            thrust::device_vector<unsigned int>& owners) {
    size_t nodes = partitions.size();
    // std::cout << "owner nodes: " << nodes << std::endl;
    thrust::fill(owners.begin(), owners.end(), 0);

    if (nodes == 1) return;
    
    // TODO just fill the first 0 0 0 owners? Then start the scan at first
    // marker. The decision wether or not to do this would be 
    /// owners.size() / nodes > X
    // for some sane X.

    SetMarkers setMarkers(owners, partitions);
    thrust::counting_iterator<unsigned int> threadIds(0);
    thrust::for_each(threadIds, threadIds + nodes, setMarkers);
    // std::cout << "markers:\n" << owners << std::endl;

    thrust::inclusive_scan(owners.begin(), owners.end(), owners.begin());

    // std::cout << "owners:\n" << owners << std::endl;
}


void DacrtNodes::ResizeUnfinished(const size_t size) {
    rayPartitions.resize(size);
    spherePartitions.resize(size);
}


std::string DacrtNodes::ToString() const {
    std::ostringstream out;
    if (UnfinishedNodes() > 0) {
        out << "Unfinished DacrtNodes:";
        for (size_t i = 0; i < UnfinishedNodes(); ++i)
            out << "\n" << i << ": " << GetUnfinished(i);
        if (DoneNodes() > 0) out << "\n";
    }
    if (DoneNodes() > 0) {
        out << "Done DacrtNodes:";
        for (size_t i = 0; i < DoneNodes(); ++i)
            out << "\n" << i << ": " << GetDone(i);
    }
    return out.str();
}

std::string DacrtNodes::ToString(RayContainer& rays, SphereContainer& spheres) const {
    std::ostringstream out;
    if (UnfinishedNodes() > 0) {
        out << "Unfinished DacrtNodes:";
        for (size_t i = 0; i < UnfinishedNodes(); ++i) {
            DacrtNode node = GetUnfinished(i);
            out << "\n" << i << ": " << node << "\n  Rays: ";
            for (unsigned int r = node.rayStart; r < node.rayEnd; ++r){
                float4 origins = *(HyperRays::GetOrigins(rays.BeginInnerRays()) + r);
                out << origins.w << ", ";
            }
            out << "\n  Spheres: ";
            for (unsigned int s = node.sphereStart; s < node.sphereEnd; ++s){
                unsigned int sphereId = *(spheres.BeginCurrentIndices() + s);
                out << sphereId << ", ";
            }
        }
        if (DoneNodes() > 0) out << "\n";
    }

    if (DoneNodes() > 0) {
        out << "Done DacrtNodes:";
        for (size_t i = 0; i < DoneNodes(); ++i) {
            DacrtNode node = GetDone(i);
            out << "\n" << i << ": " << node << "\n  Rays: ";
            for (unsigned int r = node.rayStart; r < node.rayEnd; ++r){
                float4 origins = *(HyperRays::GetOrigins(rays.BeginLeafRays()) + r);
                out << origins.w << ", ";
            }
            out << "\n  Spheres: ";
            for (unsigned int s = node.sphereStart; s < node.sphereEnd; ++s){
                unsigned int sphereId = *(spheres.BeginDoneIndices() + s);
                out << sphereId << ", ";
            }
        }
    }
    return out.str();
    
}

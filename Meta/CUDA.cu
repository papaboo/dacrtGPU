// CUDA
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <Meta/CUDA.h>

#include <iostream>

// One of these contain cutGetMaxGflopsDeviceId. No idea which (no internet)
#include <cutil.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace Meta {

hipDeviceProp_t CUDA::activeCudaDevice;

void CUDA::Initialize() {
    int deviceCount;
    if (hipGetDeviceCount(&deviceCount) == hipErrorNoDevice) {
        std::cout << "No CUDA capable device found" << std::endl;
        exit(0);
    }
 
    hipDevice_t device = cutGetMaxGflopsDeviceId();
    hipSetDevice(device);
    // cudaGLSetGLDevice(device);

    hipInit(0);

    int version;
    hipDriverGetVersion(&version);
    hipGetDeviceProperties(&activeCudaDevice, device);
    std::cout << "CUDA: version " << version/1000 << "." << version % 100 << ", using device " << std::string(activeCudaDevice.name) << std::endl;
   
}

}

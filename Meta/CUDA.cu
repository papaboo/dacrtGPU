// CUDA
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <Meta/CUDA.h>
#include <Utils.h>

// One of these contain cutGetMaxGflopsDeviceId. No idea which (no internet)
#include <cutil.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace Meta {

hipDeviceProp_t CUDA::activeCudaDevice;

void CUDA::Initialize() {
    int deviceCount;
    if (hipGetDeviceCount(&deviceCount) == hipErrorNoDevice) {
        std::cout << "No CUDA capable device found" << std::endl;
        exit(0);
    }
 
    hipDevice_t device = cutGetMaxGflopsDeviceId();
    hipSetDevice(device);
    // cudaGLSetGLDevice(device);
    CHECK_FOR_CUDA_ERROR();
    
    hipInit(0);
    CHECK_FOR_CUDA_ERROR();
    
    int version;
    hipDriverGetVersion(&version);
    hipGetDeviceProperties(&activeCudaDevice, device);
    CHECK_FOR_CUDA_ERROR();
    std::cout << "CUDA: version " << version/1000 << "." << version % 100 << ", using device " << std::string(activeCudaDevice.name) << std::endl;
   
}

}

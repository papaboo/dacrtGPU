#include "hip/hip_runtime.h"
// Shading.
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <Shading.h>

#include <Fragment.h>
#include <Meta/CUDA.h>
#include <Primitives/Sphere.h>
#include <RayContainer.h>
#include <SphereGeometry.h>
#include <Utils/Random.h>

#include <iostream>

#define PI ((float)3.14159265358979)

__global__
void ColorNormalsKernel(float4* rayOrigins,
                        float4* rayDirections,
                        unsigned int* hitIDs, // Contains information about the new rays after 
                        const Sphere* const spheres,
                        float4* emission_bounces, // result
                        const unsigned int nRays) {

    const unsigned int rayID = threadIdx.x + blockDim.x * blockIdx.x;
    if (rayID >= nRays) return;

    const unsigned int hitID = hitIDs[rayID];
    const float4 originId = rayOrigins[rayID];
    const float3 rayOrigin = make_float3(originId);
    const unsigned int fragID = originId.w;

    if (hitID == SpheresGeometry::MISSED) {
        emission_bounces[fragID] =  make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        hitIDs[rayID] = 0; // Make a note that this ray should be terminated.
        return;
    }

    const Sphere sphere = spheres[hitID];

    const float4 dir_t = rayDirections[rayID];
    const float3 dir = make_float3(dir_t);
    const float t = dir_t.w;
    
    const float3 hitPos = t * dir + rayOrigin;
    const float3 norm = normalize(hitPos - sphere.center);
    
    // Map to visible
    emission_bounces[fragID] = make_float4(norm * 0.5f + 0.5f, 1.0f);
        
    rayOrigins[rayID] = make_float4(hitPos + norm * 0.02f, fragID);

    const float3 reflectionDir = dir - norm * 2 * dot(norm, dir);
    rayDirections[rayID] = make_float4(reflectionDir, 1e30f);

    hitIDs[rayID] = hitID == 7 ? 1 : 0; // Reflect of sphere 7, just to test it
}

/**
 * Shades the fragments with the color of the normals.
 *
 * After execution the hitIDs contains 0 for rays that bounced and 0 for
 * terminated rays.
 */
void Shading::Normals(RayContainer& rays, 
                      thrust::device_vector<unsigned int>::iterator hitIDs,
                      SpheresGeometry& spheres, 
                      Fragments& frags) {

    size_t nRays = rays.EndLeafRays() - rays.BeginLeafRays();    

    struct hipFuncAttributes funcAttr;
    hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(ColorNormalsKernel));
    unsigned int blocksize = funcAttr.maxThreadsPerBlock > 256 ? 256 : funcAttr.maxThreadsPerBlock;
    unsigned int blocks = (nRays / blocksize) + 1;
    ColorNormalsKernel<<<blocks, blocksize>>>
        (RawPointer(Rays::GetOrigins(rays.BeginLeafRays())),
         RawPointer(Rays::GetDirections(rays.BeginLeafRays())),
         RawPointer(hitIDs), // Contains information about the new rays after 
         RawPointer(spheres.spheres),
         RawPointer(frags.emissionDepth), // result
         nRays);
    CHECK_FOR_CUDA_ERROR();

    rays.RemoveTerminated(hitIDs);
}    

__device__ bool d_raysTerminated;

// TODO Place all material parameters in a struct and create access methods for
// it, so I don't have to extend the shade kernels each time I add a new
// parameter.
__global__
void PathTraceKernel(float4* rayOrigins,
                     float4* rayDirections,
                     unsigned int* hitIDs, // Contains information about the new rays after 
                     const Sphere* const spheres,
                     const unsigned int* const matIDs,
                     const float4* const emission_reflections,
                     const float4* const color_refractions,
                     float4* emission_bounces, // result
                     float4* fs, // result
                     const unsigned int nRays, 
                     const unsigned int seed) {

    const unsigned int rayID = threadIdx.x + blockDim.x * blockIdx.x;
    if (rayID >= nRays) return;
    
    const unsigned int hitID = hitIDs[rayID];
    
    const float4 originId = rayOrigins[rayID];
    const float3 rayOrigin = make_float3(originId);
    const unsigned int fragID = originId.w;

    const float4 emission_bounce = emission_bounces[fragID];

    const float3 oldF = make_float3(fs[fragID]);
    if (hitID == SpheresGeometry::MISSED) {
        const float3 backgroundColor = make_float3(0.8f, 0.8f, 0.8f);
        const float3 color = oldF * backgroundColor;
        emission_bounces[fragID] = make_float4(color, emission_bounce.w);
        hitIDs[rayID] = 0; // Make a note that this ray should be terminated.
        d_raysTerminated = true;
        return;
    }
    
    const unsigned int matID = matIDs[hitID];
    const float4 emission_reflection = emission_reflections[matID];
        
    if (emission_bounce.w >= 5) {
        emission_bounces[fragID] = emission_bounce + make_float4(oldF * make_float3(emission_reflection), 0);
        hitIDs[rayID] = 0; // Make a note that this ray should be terminated.
        d_raysTerminated = true;
        return;
    }

    const float4 dir_t = rayDirections[rayID];
    float3 dir = make_float3(dir_t);
    const float t = dir_t.w;

    const Sphere sphere = spheres[hitID];

    const float3 hitPos = t * dir + rayOrigin; // we could store hitPos in the rays origin, since we should already know it from the intersection test.
    const float3 sphereNorm = normalize(hitPos - sphere.center); // The sphere's normal, pointing away from the center.
    const bool into = dot(sphereNorm, dir) < 0.0f;
    const float3 rayNorm = into ? sphereNorm : sphereNorm*-1.0f; // May flip the normal so it doesn't point away from the ray.

    Random rand = Random::Create1D(seed);
    
    float colorContribution = 0.0f;

    if (rand.NextFloat01() < emission_reflection.w) {
        // ray is reflected
        dir = dir - rayNorm * 2.0f * dot(rayNorm, dir);
    } else if (rand.NextFloat01() < color_refractions[matID].w){
        float3 reflect = dir - rayNorm * 2.0f * dot(rayNorm, dir);
        
        // Pure magic 'borrowed' from smallpt
        float nc = 1.0f, nt = 1.5f;
        float nnt = into ? nc/nt : nt/nc;
        float ddn = dot(dir, rayNorm);
        float cos2t = 1.0f - nnt * nnt * (1.0f - ddn * ddn);
        
        if (cos2t < 0.0f) {
            dir = reflect;
        } else {
            float3 tDir = normalize(dir * nnt - rayNorm * (ddn*nnt+sqrt(cos2t)));
            float a=nt-nc, b=nt+nc, R0=a*a/(b*b), c = 1-(into?-ddn : dot(tDir, sphereNorm));
            float Re=R0+(1-R0)*c*c*c*c*c;
            // float P = 0.25f + 0.5f * Re; 
            // float Tr = 1.0f - Re;
            // float RP = Re / P, TP = Tr / (1.0f-P);
            if (rand.NextFloat01() < Re) // reflection
                dir = reflect;
            else 
                dir = tDir;
        }
    } else {
    
        // ray is diffuse
        colorContribution = 1.0f;
        
        const float r1 = 2 * PI * rand.NextFloat01();
        const float r2 = rand.NextFloat01();
        const float r2s = sqrtf(r2);
        // Tangent space ?
        const float3 w = rayNorm;
        const float3 u = normalize(fabsf(w.x) > 0.1f ? 
                                   make_float3(0,1,0) : 
                                   cross(make_float3(1,0,0), w));
        const float3 v = cross(w, u);
        dir = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1.0f-r2));
    }

    const float4 color_refraction = color_refractions[matID];
    emission_bounces[fragID] = emission_bounce + make_float4(colorContribution * oldF * make_float3(emission_reflection), 1.0f);
    fs[fragID] = make_float4(oldF * make_float3(color_refraction), 0.0f);
    
    
    bool refract = dot(rayNorm, dir) < 0.0f;
    rayOrigins[rayID] = make_float4(hitPos + rayNorm * (refract ? -0.02f : 0.02f), fragID);
    rayDirections[rayID] = make_float4(dir, 1e30f);
    
    hitIDs[rayID] = 1; // Note that this ray should not be terminated. TODO
                       // Perhaps I should just use sphere missed to denote done
                       // rays and anything else to denote not done rays. Then I
                       // would save a couple of writes.
}

void Shading::Shade(RayContainer& rays, 
                    thrust::device_vector<unsigned int>::iterator hitIDs,
                    SpheresGeometry& spheres,
                    Fragments& frags) {

    size_t nRays = rays.EndLeafRays() - rays.BeginLeafRays();

    bool raysTerminated = false;
    hipMemcpyToSymbol(HIP_SYMBOL(d_raysTerminated), &raysTerminated, sizeof(bool), 0, hipMemcpyHostToDevice);

    struct hipFuncAttributes funcAttr;
    hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(PathTraceKernel));
    unsigned int blocksize = funcAttr.maxThreadsPerBlock > 256 ? 256 : funcAttr.maxThreadsPerBlock;
    unsigned int blocks = (nRays / blocksize) + 1;
    PathTraceKernel<<<blocks, blocksize>>>
        (RawPointer(Rays::GetOrigins(rays.BeginLeafRays())),
         RawPointer(Rays::GetDirections(rays.BeginLeafRays())),
         RawPointer(hitIDs), // Contains information about the new rays after 
         RawPointer(spheres.spheres),
         RawPointer(spheres.materialIDs),
         RawPointer(spheres.materials.emission_reflection),
         RawPointer(spheres.materials.color_refraction),
         RawPointer(frags.emissionDepth), // result
         RawPointer(frags.f), // result
         nRays,
         rand());
    CHECK_FOR_CUDA_ERROR();

    hipMemcpyFromSymbol(&raysTerminated, HIP_SYMBOL(d_raysTerminated), sizeof(bool));

    if (raysTerminated)
        rays.RemoveTerminated(hitIDs);
    else
        rays.ReinitLeafRays();

}

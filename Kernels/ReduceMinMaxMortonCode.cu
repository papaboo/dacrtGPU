#include "hip/hip_runtime.h"
// Kernel for reducing the min and max of a list of morton codes
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <Kernels/ReduceMinMaxMortonCode.h>

#include <Meta/CUDA.h>
#include <Utils/Morton.h>
#include <Utils/ToString.h>

#include <iostream>
#include <stdexcept>

using Utils::Morton;

namespace Kernels {
    
    __device__ 
    inline void PerformMinMaxMortonByAxisReduction(const unsigned int index, 
                                                   volatile unsigned int* min, volatile unsigned int* max,
                                                   MortonBound* bounds) {
        // Load data from global mem into registers.
        const MortonCode lhsMin = min[index * 2];
        const MortonCode rhsMin = min[index * 2 + 1];
        const MortonCode lhsMax = max[index * 2];
        const MortonCode rhsMax = max[index * 2 + 1];
        // Since the result is stored in another shared mem entry, the threads
        // need to be synced between fetching and storing.
        __syncthreads();
        
        const unsigned int axis = lhsMin & 0xE0000000;
        if (axis == lhsMin & 0xE0000000) { // Compare axis
            // Both have the same owner, so store the result back in shared
            // memory.
            min[index] = Morton::MinBy4(lhsMin.WithoutAxis(), rhsMin.WithoutAxis()) + axis;
            max[index] = Morton::MaxBy4(lhsMax.WithoutAxis(), rhsMax.WithoutAxis()) + axis;
        } else {
            // Different owners, so store the lhs in global memory and rhs in
            // shared.
            min[index] = rhsMin;
            max[index] = rhsMax;
        
            const unsigned int globalIndex = MortonCode::AxisFromCode(lhsMin);
            const MortonBound old = bounds[globalIndex];
            const unsigned int min = Morton::MinBy4(lhsMin.WithoutAxis(), old.min.WithoutAxis()) + axis;
            const unsigned int max = Morton::MaxBy4(lhsMax.WithoutAxis(), old.max.WithoutAxis()) + axis;
            bounds[globalIndex] = MortonBound::Create(min, max);
        }
    }

    __global__
    void ReduceMinMaxMortonByAxisPass2(MortonBound* intermediateBounds,
                                       const size_t intermediateBoundsSize,
                                       MortonBound* bounds,
                                       const size_t boundsSize) {
    
        __shared__ volatile unsigned int mins[128];
        __shared__ volatile unsigned int maxs[128];

        if (threadIdx.x < intermediateBoundsSize) {
            MortonBound b = intermediateBounds[threadIdx.x];
            mins[threadIdx.x] = b.min;
            maxs[threadIdx.x] = b.max;
        } else
            mins[threadIdx.x] = maxs[threadIdx.x] = MortonCode::EncodeAxis(6);
        __syncthreads();

        // Reduce 128 values left in shared memory
        if (threadIdx.x >= 64) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);

        if (threadIdx.x >= 32) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);

        if (threadIdx.x >= 16) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);

        if (threadIdx.x >= 8) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);

        if (threadIdx.x >= 4) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);
    
        if (threadIdx.x >= 2) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, mins, maxs, bounds);
    
        // Reduce the last value, comparing it to what is already stored in bounds
        const unsigned int lhsMin = mins[0];
        const unsigned int lhsMax = maxs[0];
        const unsigned int axis = lhsMin & 0xE0000000;
        const unsigned int globalIndex = MortonCode::AxisFromCode(lhsMin);
        if (globalIndex != 6) { // If any dummy values (axis == 6) have been used,
            // then now is the time to discard them.
            const MortonBound old = bounds[globalIndex];
            const unsigned int min = Morton::MinBy4(lhsMin & 0x1FFFFFFF, old.min & 0x1FFFFFFF) + axis;
            const unsigned int max = Morton::MaxBy4(lhsMax & 0x1FFFFFFF, old.max & 0x1FFFFFFF) + axis;
            bounds[globalIndex] = MortonBound::Create(min, max);
        }
    }

    __global__
    void ReduceMinMaxMortonByAxisPass1(const unsigned int* const mortonCodes,
                                       const size_t inputSize,
                                       MortonBound* intermediateBounds,
                                       MortonBound* bounds,
                                       const size_t boundsSize) {

        // Fill bounds with default values
        for (unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; i < boundsSize; i += blockDim.x * gridDim.x)
            bounds[i] = MortonBound::Create(0xDFFFFFFF, 0xC0000000);

        __shared__ volatile unsigned int min[256];
        __shared__ volatile unsigned int max[256];
    
        const size_t beginIndex = inputSize * blockIdx.x / gridDim.x;
        const size_t endIndex = inputSize * (blockIdx.x + 1) / gridDim.x;
    
        size_t currentIndex = beginIndex + threadIdx.x;

        // Fill initial values
        min[threadIdx.x] = max[threadIdx.x] = currentIndex < endIndex ? MortonCode(mortonCodes[currentIndex]) : MortonCode::EncodeAxis(6);
        currentIndex += blockDim.x;
        __syncthreads();
    
        // While still values left, load them and perform reduction
        while (currentIndex < endIndex) {
            // Fetch new data from global mem to shared mem
            unsigned int code = currentIndex < endIndex ? MortonCode(mortonCodes[currentIndex]) : MortonCode::EncodeAxis(6);
            min[blockDim.x + threadIdx.x] = max[blockDim.x + threadIdx.x] = code;
            __syncthreads();

            PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);
        }
        __syncthreads();

        // Reduce 128 values left in shared memory
        if (threadIdx.x >= 64) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);

        if (threadIdx.x >= 32) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);

        if (threadIdx.x >= 16) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);

        if (threadIdx.x >= 8) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);

        if (threadIdx.x >= 4) return;
        PerformMinMaxMortonByAxisReduction(threadIdx.x, min, max, bounds);
    
        if (threadIdx.x >= 2) return;
        // Store the last 2 values, since they may overlap with the first and last
        // value in neighbouring blocks.
        intermediateBounds[2 * blockIdx.x + threadIdx.x] = MortonBound::Create((unsigned int)min[threadIdx.x], 
                                                                               (unsigned int)max[threadIdx.x]);
    }


    void ReduceMinMaxMortonByAxis(thrust::device_vector<unsigned int>::iterator mortonBegin,
                                  thrust::device_vector<unsigned int>::iterator mortonEnd,
                                  thrust::device_vector<MortonBound>::iterator boundsBegin,
                                  thrust::device_vector<MortonBound>::iterator boundsEnd) {

        // Verify that CUDA is initialized
        if (!Meta::CUDA::initialized)
            throw std::runtime_error("CUDA wasn't initialized. Can't lookup kernel properties");
    
        const size_t inputSize = mortonEnd - mortonBegin;
        const size_t boundsSize = boundsEnd - boundsBegin; // Always 6, but this looks less like magic.
    
        // struct hipFuncAttributes funcAttr;
        // hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(ReduceMinMaxMortonByAxisPass1));
        // const unsigned int blockDim = funcAttr.maxThreadsPerBlock > 128 ? 128 : funcAttr.maxThreadsPerBlock;
        const unsigned int blockDim = 128;
        const unsigned int blocks = Meta::CUDA::activeCudaDevice.multiProcessorCount;

        static thrust::device_vector<MortonBound> intermediateBounds(boundsSize * 2);
    
        ReduceMinMaxMortonByAxisPass1<<<blocks, blockDim>>>(RawPointer(mortonBegin), inputSize,
                                                            RawPointer(intermediateBounds),
                                                            thrust::raw_pointer_cast(&*boundsBegin), boundsSize);

        // std::cout << "intermediateBounds\n" << intermediateBounds << std::endl;
    
        // std::cout << "bounds:\n";
        // for (int i = 0; i < boundsSize; ++i) {
        //     std::cout << i << ": " << boundsBegin[i];
        //     if (i < boundsSize-1)
        //         std::cout << "\n";
        // }
        // std::cout << std::endl;
    
        ReduceMinMaxMortonByAxisPass2<<<1, 128>>>(RawPointer(intermediateBounds), intermediateBounds.size(),
                                                  thrust::raw_pointer_cast(&*boundsBegin), boundsSize);

    }

}

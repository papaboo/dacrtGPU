#include "hip/hip_runtime.h"
// Hyper ray abstraction
// -----------------------------------------------------------------------------
// Copyright (C) 2012, See authors
//
// This program is open source and distributed under the New BSD License. See
// license for more detail.
// -----------------------------------------------------------------------------

#include <Rays.h>

#include <Meta/CUDA.h>
#include <Utils/Random.h>
#include <Utils/ToString.h>

#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <ostream>
#include <iomanip>


inline float2 RandomFloat2() {
    float x = (float)rand() / (float)RAND_MAX;
    float y = (float)rand() / (float)RAND_MAX;
    return make_float2(x, y);
}

__constant__ int d_width;
__constant__ int d_height;
__constant__ int d_sqrtSamples;
__constant__ float3 d_cx;
__constant__ float3 d_cy;

struct DeviceCreateRays {

    DeviceCreateRays(const int width, const int height, const int sqrtSamples,
                     const float3& cx, const float3& cy) {
        hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_sqrtSamples), &sqrtSamples, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_cx), &cx, sizeof(float3));
        hipMemcpyToSymbol(HIP_SYMBOL(d_cy), &cy, sizeof(float3));
    }
    
    __host__ __device__
    thrust::tuple<float4, float4> operator()(const float2 rand, const unsigned int index) {
        // Reverse: int index = (x + y * width) * samples + subX + subY * sqrtSamples;
        const unsigned short subX = index % d_sqrtSamples;
        const unsigned short subY = (index / d_sqrtSamples) % d_sqrtSamples;
        const int lowResIndex = index / (d_sqrtSamples * d_sqrtSamples);
        const unsigned short x = lowResIndex % d_width;
        const unsigned short y = lowResIndex / d_width;
    
        const float3 camOrigin = make_float3(50.0f, 52.0f, 295.6f);
        const float3 camDir = make_float3(0.0f, -0.042612f,-1.0f);

        const float r1 = 2.0f * rand.x;
        const float dx = r1 < 1.0f ? sqrt(r1) - 1.0f : 1.0f - sqrt(2.0f - r1);
        const float r2 = 2.0f * rand.y;
        const float dy = r2 < 1.0f ? sqrt(r2) - 1.0f : 1.0f - sqrt(2.0f - r2);
        
        const float3 rayDir = d_cx * (((subX + 0.5f + dx) / d_sqrtSamples + x) / d_width - 0.5f) 
            + d_cy * (((subY + 0.5f + dy) / d_sqrtSamples + y) / d_height - 0.5f) + camDir;
        
        
        const float3 rayOrigin = camOrigin + 130.0f * rayDir;

        return thrust::tuple<float4, float4>(make_float4(rayOrigin, index),
                                             make_float4(normalize(rayDir), 0.0f));
    }
};

// TODO Adopted from a linearly indexed thrust operator. Could probably be speed
// up as a 2D kernel.
__global__
void CreateRaysKernel(const int width, const int height, const int sqrtSamples,
                      const float3 cx, const float3 cy,
                      const unsigned int seed,
                      float4* origins, float4* directions) {

    const unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= width * height * sqrtSamples * sqrtSamples) return;

    // Reverse: int index = (x + y * width) * samples + subX + subY * sqrtSamples;
    const unsigned short subX = index % sqrtSamples;
    const unsigned short subY = (index / sqrtSamples) % sqrtSamples;
    const int lowResIndex = index / (sqrtSamples * sqrtSamples);
    const unsigned short x = lowResIndex % width;
    const unsigned short y = lowResIndex / width;
    
    const float3 camOrigin = make_float3(50.0f, 52.0f, 295.6f);
    const float3 camDir = make_float3(0.0f, -0.042612f,-1.0f);

    Random rand = Random::Create1D(seed);

    const float r1 = 2.0f * rand.NextFloat01();
    const float dx = r1 < 1.0f ? sqrt(r1) - 1.0f : 1.0f - sqrt(2.0f - r1);
    const float r2 = 2.0f * rand.NextFloat01();
    const float dy = r2 < 1.0f ? sqrt(r2) - 1.0f : 1.0f - sqrt(2.0f - r2);
    
    const float3 rayDir = cx * (((subX + 0.5f + dx) / sqrtSamples + x) / width - 0.5f) 
        + cy * (((subY + 0.5f + dy) / sqrtSamples + y) / height - 0.5f) + camDir;
    
    const float3 rayOrigin = camOrigin + 130.0f * rayDir;
    origins[index] = make_float4(rayOrigin, index);
    directions[index] = make_float4(normalize(rayDir), 0.0f);
}

Rays::Rays(const int width, const int height, const int sqrtSamples) {
    const int size = width * height * sqrtSamples * sqrtSamples;
    origins = thrust::device_vector<float4>(size);
    axisUVs = thrust::device_vector<float4>(size);

    const float3 camDir = make_float3(0.0f, -0.042612f,-1.0f);
    
    const float3 cx = make_float3(width * 0.5135f / height, 0, 0);
    const float3 cy = normalize(cross(cx, camDir)) * 0.5135f;

    bool useHostRandom = false;
    if (useHostRandom) {
        // Generate random numbers
        thrust::host_vector<float2> host_random(size);
        thrust::generate(host_random.begin(), host_random.end(), RandomFloat2);
        thrust::device_vector<float2> random = host_random;
        
        DeviceCreateRays deviceCreateRays(width, height, sqrtSamples, cx, cy);
        thrust::transform(random.begin(), random.end(), thrust::counting_iterator<unsigned int>(0), 
                          Begin(), deviceCreateRays);
    } else {
        struct hipFuncAttributes funcAttr;
        hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(CreateRaysKernel));
        unsigned int blocksize = funcAttr.maxThreadsPerBlock > 256 ? 256 : funcAttr.maxThreadsPerBlock;
        unsigned int blocks = (size / blocksize) + 1;
        std::cout << "CreateRaysKernel<<<" << blocks << ", " << blocksize << ">>>" << std::endl;
        CreateRaysKernel<<<blocks, blocksize>>>(width, height, sqrtSamples, cx, cy, rand(),
                                                RawPointer(origins), RawPointer(axisUVs));
        CHECK_FOR_CUDA_ERROR();
    }

    representation = RayRepresentation;

    std::cout << "Rays:" << std::endl;
    for (int i = 0; i < 10; ++i)
        std::cout << i << ": " << GetAsRay(i+size/2) << std::endl;
}

struct RaysToHyperRays {
    __host__ __device__
    inline float4 operator()(const float4& direction) const {
        const float3 axisUV = HyperRay::DirectionToAxisUV(make_float3(direction));
        return make_float4(axisUV, direction.w);
    }
};

struct HyperRaysToRays {
    __host__ __device__
    inline float4 operator()(const float4& axisUV) const {
        const float3 dir = normalize(HyperRay::AxisUVToDirection(make_float3(axisUV)));
        return make_float4(dir, axisUV.w);
    }
};

void Rays::Convert(const Representation r) {
    if (representation == r) return;

    if (Size() > 0) {
        // std::cout << "Convert from " << representation << " to " << r << std::endl;
        switch(representation) {
        case RayRepresentation: 
            switch (r) {
            case HyperRayRepresentation:
                thrust::transform(axisUVs.begin(), axisUVs.end(), axisUVs.begin(), RaysToHyperRays());
                break;
            default:
                std::cout << "No conversion from " << representation << " to " << r << std::endl;
                return;
            }
            break;
        case HyperRayRepresentation:
            switch (r) {
            case RayRepresentation:
                thrust::transform(axisUVs.begin(), axisUVs.end(), axisUVs.begin(), HyperRaysToRays());
                break;
            default:
                std::cout << "No conversion from " << representation << " to " << r << std::endl;
                return;
            }
            break;
        default:
            std::cout << "Converting from " << representation << " not supported." << std::endl;
            return;
        }
    }

    // std::cout << "Converted from " << representation << " to " << r << std::endl;
    
    representation = r;
}

std::string Rays::ToString() const {
    std::ostringstream out;
    out << "HyperRays";
    if (representation == RayRepresentation)
        for (size_t i = 0; i < Size(); ++i)
            out << "\n" << GetAsRay(i);
    
    else if (representation == HyperRayRepresentation)
        for (size_t i = 0; i < Size(); ++i)
            out << "\n" << GetAsHyperRay(i);
    return out.str();
}
